#include "hip/hip_runtime.h"
#include "mySolver.h"
#include <iostream>

#include <stdio.h>

#include <thrust/version.h>
#include <cusp/version.h>
#include <cusp/csr_matrix.h>
#include <cusp/print.h>
//#include <cusp/krylov/cg.h>
#include <cusp/krylov/bicgstab.h>

//#include <cusp/precond/smoothed_aggregation.h>
//#include <cusp/precond/ainv.h>

#include "hip/hip_runtime.h"
#include ""

//Hide Cuda Api calls
hipError_t addWithCuda_(int *c, const int *a, const int *b, size_t size);
__global__ void addKernel_(int *c, const int *a, const int *b);


CUDASOLVER_EXPORT mySolver::mySolver()
{
	myMatrix = new cusp::csr_matrix<int,float,cusp::device_memory>();
}

CUDASOLVER_EXPORT mySolver::~mySolver()
{
	delete myMatrix;
}


void CUDASOLVER_EXPORT mySolver::setMatrix( cpuCSRMatrix & mat )
{
	delete myMatrix;
	myMatrix = new cusp::csr_matrix<int,float,cusp::device_memory>(mat.getn(), mat.getm(), mat.geta().size());
	updateMatrix(mat);

	//cusp::print(*myMatrix);
}


void CUDASOLVER_EXPORT mySolver::updateMatrix(cpuCSRMatrix & mat)
{
	myMatrix->column_indices = cusp::array1d<float, cusp::host_memory>(mat.getja().begin(), mat.getja().end());
	myMatrix->row_offsets = cusp::array1d<float, cusp::host_memory>(mat.getia().begin(), mat.getia().end());
	myMatrix->values = cusp::array1d<float, cusp::host_memory>(mat.geta().begin(), mat.geta().end());

}



void CUDASOLVER_EXPORT mySolver::solve( floatVector & x, floatVector & b )
{
	cusp::csr_matrix<int,float,cusp::device_memory> & A = *myMatrix;

	// allocate storage for solution (x) and right hand side (b)
    cusp::array1d<float, cusp::device_memory> x_(x.size(), 0);//x_(x.begin(), x.end());
	cusp::array1d<float, cusp::device_memory> b_(b.begin(), b.end());

    // set stopping criteria:
    //  iteration_limit    = 100
    //  relative_tolerance = 1e-3
    cusp::convergence_monitor<float> monitor(b_, 1000, 1e-6);

    // set preconditioner (identity)
    cusp::identity_operator<float, cusp::device_memory> M(A.num_rows, A.num_rows);
	//cusp::precond::scaled_bridson_ainv<float, cusp::device_memory> M(A, .1);
	//cusp::precond::smoothed_aggregation<int, float, cusp::device_memory> M(A);

    // solve the linear system A * x = b with the Conjugate Gradient method
   // cusp::krylov::cg(A, x_, b_, monitor, M);
	cusp::krylov::bicgstab(A, x_, b_, monitor, M);
	
	monitor.print();


	//looks frigging inefficient
	for(unsigned int i = 0; i < x.size(); i++){
		x[i] = x_[i];
	}

	//cusp::print(b_);
}


void CUDASOLVER_EXPORT mySolver::checkMatrix( cpuCSRMatrix & mat )
{

}


int  CUDASOLVER_EXPORT mySolver::main()
{

	int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;

    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;

    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;

    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
	std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;

	cusp::csr_matrix<int,float,cusp::host_memory> A(4,3,6);

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
	

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda_(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "Now, enter a number and press Enter\n";
	int anything;
	std::cin >> anything;

    return 0;
}

__global__ void addKernel_(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda_(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel_<<<1, size>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
