#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include <stdio.h>

#include <thrust/version.h>
#include <cusp/version.h>
#include <cusp/csr_matrix.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int  CUDASOLVER_EXPORT CUDA_STUFF::main()
{

	int cuda_major =  CUDA_VERSION / 1000;
    int cuda_minor = (CUDA_VERSION % 1000) / 10;

    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;

    int cusp_major = CUSP_MAJOR_VERSION;
    int cusp_minor = CUSP_MINOR_VERSION;

    std::cout << "CUDA   v" << cuda_major   << "." << cuda_minor   << std::endl;
    std::cout << "Thrust v" << thrust_major << "." << thrust_minor << std::endl;
	std::cout << "Cusp   v" << cusp_major   << "." << cusp_minor   << std::endl;

	cusp::csr_matrix<int,float,cusp::host_memory> A(4,3,6);

    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
	

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	std::cout << "Now, enter a number and press Enter\n";
	int anything;
	std::cin >> anything;

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
